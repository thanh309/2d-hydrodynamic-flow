// #include <math.h>

#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>


// Constants and parameters
const int scale = 2;
const double length_x = 1.0;
const double length_y = 2.0;
const int nx = 50 * scale;
const int ny = 100 * scale;
const double dx = length_x / nx;
const double dy = length_y / ny;
const double Re = 0.05;
const double U = 1.0;
const double L = 1.0;
const double nu = U * L / Re;
const double dt = 0.000001;
const double div_tolerance = 1e-3;
const double beta0 = 1.0;
const int max_iterations = 200;
const double total_time = 0.001;
const double beta = beta0 / (2*dt*(1 / (dx*dx) + 1 / (dy*dy)));

const int max_number_of_frames = 50;

// Inlet and outlet positions
const int inlet_start = scale * (44 - 2);
const int inlet_end = scale * (56 + 2);
const int outlet1_start = scale * (10 - 2);
const int outlet1_end = scale * (23 + 2);
const int outlet2_start = scale * (44 - 2);
const int outlet2_end = scale * (56 + 2);
const int outlet3_start = scale * (77 - 2);
const int outlet3_end = scale * (90 + 2);

#define s_d sizeof(double)
#define s_i sizeof(int)
#define s_i16 sizeof(uint16_t)

// CUDA error checking macro
#define CUDA_CHECK_ERROR(call)                                             \
    {                                                                      \
        hipError_t err = call;                                            \
        if (err != hipSuccess) {                                          \
            fprintf(stderr, "CUDA error in %s at line %d: %s\n", __FILE__, \
                    __LINE__, hipGetErrorString(err));                    \
            exit(EXIT_FAILURE);                                            \
        }                                                                  \
    }

// Host velocity fields, pressure, and divergence
double **h_u;                   // x-velocity on cell faces (ny x nx+1)
double **h_v;                   // y-velocity on cell faces (ny+1 x nx)
double **h_p;                   // pressure at cell centers (ny x nx)
double **h_div;                 // divergence (ny x nx)
double **h_u_center;            // x-velocity at cell centers (ny x nx)
double **h_v_center;            // y-velocity at cell centers (ny x nx)
double **h_velocity_magnitude;  // Velocity magnitude (ny x nx)
int *h_converged;               // Convergence flag

// Device 1D arrays
double *d_u;                   // x-velocity on cell faces
double *d_v;                   // y-velocity on cell faces
double *d_p;                   // pressure at cell centers
double *d_div;                 // divergence
double *d_u_next;              // temporary x-velocity
double *d_v_next;              // temporary y-velocity
double *d_u_center;            // x-velocity at cell centers
double *d_v_center;            // y-velocity at cell centers
double *d_velocity_magnitude;  // Velocity magnitude
int *d_converged;              // Convergence flag

// Function to convert float to half-precision float
uint16_t float_to_half(float f) {
    // Convert float to bits
    uint32_t f_bits = *((uint32_t *)&f);

    // Extract sign, mantissa and exponent
    uint16_t sign = (f_bits >> 31) & 0x01;
    uint16_t exp = (f_bits >> 23) & 0xFF;
    uint32_t mant = f_bits & 0x7FFFFF;

    // Handle special cases
    if (f == 0.0f) return 0;
    if (exp == 0xFF && mant == 0) return (sign << 15) | 0x7C00;  // Infinity
    if (exp == 0xFF && mant != 0) return (sign << 15) | 0x7E00;  // NaN

    // Calculate exponent for half precision
    int16_t half_exp = exp - 127 + 15;

    // Handle normal numbers
    if (half_exp >= 1 && half_exp <= 30) {
        // Normal number
        mant = mant >> 13;
        return (sign << 15) | (half_exp << 10) | mant;
    } else if (half_exp <= 0 && half_exp >= -10) {
        // Subnormal number (denormalized)
        // Shift mantissa to account for the exponent difference
        // Include the implied leading 1 bit for the mantissa
        mant = (mant | 0x800000) >> (14 - half_exp);
        return (sign << 15) | mant;
    } else if (half_exp > 30) {
        // Overflow to infinity
        return (sign << 15) | 0x7C00;
    } else {
        // Underflow to zero
        return (sign << 15);
    }
}

// Function to allocate memory for 2D arrays on host
double **allocate_2d_array(int rows, int cols) {
    double **arr = (double **)malloc(rows*sizeof(double *));
    double *data = (double *)calloc(rows*cols, s_d);
    for (int i = 0; i < rows; i++) {
        arr[i] = &data[i*cols];
    }
    return arr;
}

// Function to free memory for 2D arrays on host
void free_2d_array(double **arr) {
    free(arr[0]);  // Free the data block
    free(arr);     // Free the pointers
}

// Function to copy 2D host arrays to 1D device arrays
void copy_host_to_device() {
    // Copy u
    for (int j = 0; j < ny; j++) {
        CUDA_CHECK_ERROR(hipMemcpy(d_u + j*(nx+1), h_u[j], (nx+1)*s_d,
                                    hipMemcpyHostToDevice));
    }

    // Copy v
    for (int j = 0; j < ny+1; j++) {
        CUDA_CHECK_ERROR(hipMemcpy(d_v + j*nx, h_v[j], nx*s_d,
                                    hipMemcpyHostToDevice));
    }

    // Copy p
    for (int j = 0; j < ny; j++) {
        CUDA_CHECK_ERROR(hipMemcpy(d_p + j*nx, h_p[j], nx*s_d,
                                    hipMemcpyHostToDevice));
    }
}

// Function to copy 1D device arrays to 2D host arrays
void copy_device_to_host() {
    // Copy center velocities and magnitude
    for (int j = 0; j < ny; j++) {
        CUDA_CHECK_ERROR(hipMemcpy(h_u_center[j], d_u_center + j*nx,
                                    nx*s_d,
                                    hipMemcpyDeviceToHost));
        CUDA_CHECK_ERROR(hipMemcpy(h_v_center[j], d_v_center + j*nx,
                                    nx*s_d,
                                    hipMemcpyDeviceToHost));
        CUDA_CHECK_ERROR(hipMemcpy(h_velocity_magnitude[j], d_velocity_magnitude + j*nx,
                                    nx*s_d,
                                    hipMemcpyDeviceToHost));
    }
}

// CUDA kernel to apply boundary conditions
__global__ void apply_boundary_conditions_kernel(double *u, double *v,
                                                 int nx, int ny, int inlet_start, int inlet_end,
                                                 int outlet1_start, int outlet1_end, int outlet2_start,
                                                 int outlet2_end, int outlet3_start, int outlet3_end) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    // Left boundary (inlet): u = const, v = 0 at inlet; u = 0, v = 0 elsewhere
    if (i == 0 && j < ny) {
        if (j >= inlet_start && j <= inlet_end) {
            u[j*(nx+1) + 0] = 0.1; // Inlet velocity
        } else {
            u[j*(nx+1) + 0] = 0.0; // No-slip on rest of the left wall
        }
    }
    if (i == 0 && j <= ny) {
        v[j*nx + 0] = 0.0; // No y-velocity at left boundary
    }

    // Right boundary (outlets): zero-gradient for u at outlets; u = 0 elsewhere
    if (i == nx && j < ny) {
        if ((j >= outlet1_start && j <= outlet1_end) ||
            (j >= outlet2_start && j <= outlet2_end) ||
            (j >= outlet3_start && j <= outlet3_end)) {
            u[j*(nx+1) + nx] = u[j*(nx+1) + nx-1]; // Zero-gradient at outlets
        } else {
            u[j*(nx+1) + nx] = 0.0; // No-slip on rest of the right wall
        }
    }
    if (i == nx-1 && j <= ny) {
        if (j > 0 && j < ny) {
            v[j*nx + nx-1] = v[j*nx + nx-2]; // Zero-gradient for v at right boundary
        } else {
            v[j*nx + nx-1] = 0.0;
        }
    }

    // Top and bottom boundaries: no-slip condition
    if (j == 0 && i <= nx) {
        u[0*(nx+1) + i] = 0.0;     // Bottom
    }
    if (j == ny-1 && i <= nx) {
        u[(ny-1)*(nx+1) + i] = 0.0;  // Top
    }
    if (j == 0 && i < nx) {
        v[0*nx + i] = 0.0;     // Bottom
    }
    if (j == ny && i < nx) {
        v[ny*nx + i] = 0.0;    // Top
    }

    // Velocity clamping to prevent numerical instability
    if (i <= nx && j < ny) {
        if (u[j*(nx+1) + i] > 5.0) u[j*(nx+1) + i] = 5.0;
        if (u[j*(nx+1) + i] < -5.0) u[j*(nx+1) + i] = -5.0;
    }

    if (i < nx && j <= ny) {
        if (v[j*nx + i] > 5.0) v[j*nx + i] = 5.0;
        if (v[j*nx + i] < -5.0) v[j*nx + i] = -5.0;
    }
}

// CUDA kernel to compute the divergence in each cell
__global__ void compute_divergence_kernel(double *u, double *v, double *div,
                                          int nx, int ny, double dx,
                                          double dy) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        div[j*nx + i] = (u[j*(nx+1) + i+1] - u[j*(nx+1) + i]) / dx +
                          (v[(j+1)*nx + i] - v[j*nx + i]) / dy;
    }
}

// CUDA kernel to update velocities based on Navier-Stokes equation
__global__ void update_velocities_kernel(double *u, double *v, double *p,
                                         double *u_next, double *v_next, int nx,
                                         int ny, double dx, double dy,
                                         double dt, double nu) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    // Internal cells for u (excluding boundaries)
    if (i >= 1 && i < nx && j >= 1 && j < ny-1) {
        // Calculate averages for u at cell centers
        double u_i__j = 0.5*(u[j*(nx+1) + i-1] + u[j*(nx+1) + i]);
        double u_i_plus1__j =
            0.5*(u[j*(nx+1) + i] + u[j*(nx+1) + i+1]);

        // Calculate uv product terms (Eq. uv-average)
        double uv_i_plushalf__j_minushalf =
            0.25*(u[(j-1)*(nx+1) + i] + u[j*(nx+1) + i]) *
            (v[j*nx + i-1] + v[j*nx + i]);
        double uv_i_plushalf__j_plushalf =
            0.25*(u[j*(nx+1) + i] + u[(j+1)*(nx+1) + i]) *
            (v[(j+1)*nx + i-1] + v[(j+1)*nx + i]);

        // Implement u-momentum equation (Eq. u-finite-diff)
        u_next[j*(nx+1) + i] =
            u[j*(nx+1) + i] + dt * (
                - (u_i_plus1__j*u_i_plus1__j - u_i__j*u_i__j) / dx
                - (uv_i_plushalf__j_plushalf - uv_i_plushalf__j_minushalf) / dy +
                - (p[j*nx + i] - p[j*nx + i-1]) / dx
                + nu*(
                    (u[j*(nx+1) + i+1] - 2*u[j*(nx+1) + i] + u[j*(nx+1) + i-1]) / (dx*dx)
                    + (u[(j+1)*(nx+1) + i] - 2*u[j*(nx+1) + i] + u[(j-1)*(nx+1) + i]) / (dy*dy)
                )
            );
    }

    // Internal cells for v (excluding boundaries)
    if (i >= 1 && i < nx-1 && j >= 1 && j < ny) {
        // Calculate averages for v at cell centers
        double v_i__j = 0.5*(v[(j-1)*nx + i] + v[j*nx + i]);
        double v_i__j_plus1 = 0.5*(v[j*nx + i] + v[(j+1)*nx + i]);

        // Calculate uv product terms (Eq. uv-average)
        double uv_i_minushalf__j_plushalf =
            0.25*(u[(j-1)*(nx+1) + i] + u[j*(nx+1) + i]) *
            (v[j*nx + i-1] + v[j*nx + i]);
        double uv_i_plushalf__j_plushalf =
            0.25*(u[(j-1)*(nx+1) + i+1] + u[j*(nx+1) + i+1]) *
            (v[j*nx + i] + v[j*nx + i+1]);

        // Implement v-momentum equation (Eq. v-finite-diff)
        v_next[j*nx + i] =
            v[j*nx + i] + dt * (
                - (v_i__j_plus1*v_i__j_plus1 - v_i__j*v_i__j) / dy
                - (uv_i_plushalf__j_plushalf - uv_i_minushalf__j_plushalf) / dx +
                - (p[j*nx + i] - p[(j-1)*nx + i]) / dy
                + nu*(
                    (v[j*nx + i+1] - 2*v[j*nx + i] + v[j*nx + i-1]) / (dx*dx)
                    + (v[(j+1)*nx + i] - 2*v[j*nx + i] + v[(j-1)*nx + i]) / (dy*dy)
                )
            );
    }
}

// CUDA kernel to copy the updated velocities back to the main velocity arrays
__global__ void update_velocities_final_kernel(double *u, double *v,
                                               double *u_next, double *v_next,
                                               int nx, int ny) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i >= 1 && i < nx && j >= 1 && j < ny-1) {
        u[j*(nx+1) + i] = u_next[j*(nx+1) + i];
    }

    if (i >= 1 && i < nx-1 && j >= 1 && j < ny) {
        v[j*nx + i] = v_next[j*nx + i];
    }
}

// CUDA kernel to perform pressure correction iterations (Red-Black - Red Phase)
__global__ void pressure_correction_kernel_red(double *u, double *v, double *p,
                                            double *div, int nx,
                                            int ny, double dx, double dy,
                                            double dt, double beta,
                                            double div_tolerance,
                                            int *converged) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i < nx && j < ny && (i + j) % 2 == 0) { // Process only red cells
        if (fabs(div[j*nx + i]) > div_tolerance) {
            atomicAnd(converged, 0);

            double delta_p = -beta*div[j*nx + i];
            p[j*nx + i] += delta_p * 0.7;

            // Adjust velocity components (direct assignment for red phase)
            if (i+1 <= nx) u[j*(nx+1) + i+1] += 0.5*dt/dx * delta_p;
            if (i >= 0) u[j*(nx+1) + i] -= 0.5*dt/dx * delta_p;
            if (j+1 <= ny) v[(j+1)*nx + i] += 0.5*dt/dy * delta_p;
            if (j >= 0) v[j*nx + i] -= 0.5*dt/dy * delta_p;
        }
    }
}

// CUDA kernel to perform pressure correction iterations (Red-Black - Black Phase)
__global__ void pressure_correction_kernel_black(double *u, double *v, double *p,
                                            double *div, int nx,
                                            int ny, double dx, double dy,
                                            double dt, double beta,
                                            double div_tolerance,
                                            int *converged) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i < nx && j < ny && (i + j) % 2 != 0) { // Process only black cells
        if (fabs(div[j*nx + i]) > div_tolerance) {
            atomicAnd(converged, 0);

            double delta_p = -beta*div[j*nx + i];
            p[j*nx + i] += delta_p * 0.7;

            // Adjust velocity components (direct assignment for black phase)
            if (i+1 <= nx) u[j*(nx+1) + i+1] += 0.5*dt/dx * delta_p;
            if (i >= 0) u[j*(nx+1) + i] -= 0.5*dt/dx * delta_p;
            if (j+1 <= ny) v[(j+1)*nx + i] += 0.5*dt/dy * delta_p;
            if (j >= 0) v[j*nx + i] -= 0.5*dt/dy * delta_p;
        }
    }
}

// CUDA kernel to calculate velocities at cell centers and magnitude
__global__ void calculate_center_velocities_and_magnitude_kernel(
    double *u, double *v, double *u_center, double *v_center,
    double *velocity_magnitude, int nx, int ny) {
    int i = blockIdx.x*blockDim.x + threadIdx.x;
    int j = blockIdx.y*blockDim.y + threadIdx.y;

    if (i < nx && j < ny) {
        u_center[j*nx + i] = 0.5*(u[j*(nx+1) + i] + u[j*(nx+1) + i+1]);
        v_center[j*nx + i] = 0.5*(v[j*nx + i] + v[(j+1)*nx + i]);
        velocity_magnitude[j*nx + i] =
            sqrt(u_center[j*nx + i]*u_center[j*nx + i] +
                 v_center[j*nx + i]*v_center[j*nx + i]);
    }
}

// Function to write the current state to the binary file
void write_state_to_binary(FILE *fp_u_center, FILE *fp_v_center,
                           FILE *fp_magnitude) {
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            uint16_t half_val = float_to_half((float)h_u_center[j][i]);
            fwrite(&half_val, s_i16, 1, fp_u_center);
        }
    }

    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            uint16_t half_val = float_to_half((float)h_v_center[j][i]);
            fwrite(&half_val, s_i16, 1, fp_v_center);
        }
    }

    for (int j = 0; j < ny; j++) {
        for (int i = 0; i < nx; i++) {
            uint16_t half_val =
                float_to_half((float)h_velocity_magnitude[j][i]);
            fwrite(&half_val, s_i16, 1, fp_magnitude);
        }
    }
}

// Function to write metadata to a JSON file
void write_metadata_to_json(const char *filename, double compute_time_seconds, double total_time_seconds) {
    FILE *fp = fopen(filename, "w");
    if (fp == NULL) {
        perror("Error opening file");
        return;
    }

    fprintf(fp, "{\n");
    fprintf(fp, "    \"length_x\": %f,\n", length_x);
    fprintf(fp, "    \"length_y\": %f,\n", length_y);
    fprintf(fp, "    \"nx\": %d,\n", nx);
    fprintf(fp, "    \"ny\": %d,\n", ny);
    fprintf(fp, "    \"dx\": %f,\n", dx);
    fprintf(fp, "    \"dy\": %f,\n", dy);
    fprintf(fp, "    \"dt\": %f,\n", dt);
    fprintf(fp, "    \"nu\": %f,\n", nu);
    fprintf(fp, "    \"Re\": %f,\n", Re);
    fprintf(fp, "    \"div_tolerance\": %e,\n", div_tolerance);
    fprintf(fp, "    \"beta0\": %f,\n", beta0);
    fprintf(fp, "    \"max_iterations\": %d,\n", max_iterations);
    fprintf(fp, "    \"total_time\": %f,\n", total_time);
    fprintf(fp, "    \"beta\": %f,\n", beta);
    fprintf(fp, "    \"inlet_start_j\": %d,\n", inlet_start);
    fprintf(fp, "    \"inlet_end_j\": %d,\n", inlet_end);
    fprintf(fp, "    \"outlet1_start_j\": %d,\n", outlet1_start);
    fprintf(fp, "    \"outlet1_end_j\": %d,\n", outlet1_end);
    fprintf(fp, "    \"outlet2_start_j\": %d,\n", outlet2_start);
    fprintf(fp, "    \"outlet2_end_j\": %d,\n", outlet2_end);
    fprintf(fp, "    \"outlet3_start_j\": %d,\n", outlet3_start);
    fprintf(fp, "    \"outlet3_end_j\": %d,\n", outlet3_end);
    fprintf(fp, "    \"data_dtype\": \"float16\",\n");
    fprintf(fp, "    \"output_interval_in_c_steps\": %d,\n",
            (int)(total_time / dt) / max_number_of_frames);
    fprintf(fp, "    \"num_frames_output\": %d,\n",
            ((int)(total_time / dt) /
             ((int)(total_time / dt) / max_number_of_frames)));
    fprintf(fp, "    \"total_compute_time_seconds\": %f,\n",
            compute_time_seconds);
    fprintf(fp, "    \"total_time_seconds\": %f,\n",
            total_time_seconds);
    fprintf(fp, "    \"parallelization\": \"CUDA_RedBlack\"\n");

    fprintf(fp, "}");
    fclose(fp);
}

int main() {
    clock_t total_start_time = clock();
    double total_compute_time_seconds = 0.0;

    // Allocate host memory
    h_u = allocate_2d_array(ny, nx+1);
    h_v = allocate_2d_array(ny+1, nx);
    h_p = allocate_2d_array(ny, nx);
    h_div = allocate_2d_array(ny, nx);
    h_u_center = allocate_2d_array(ny, nx);
    h_v_center = allocate_2d_array(ny, nx);
    h_velocity_magnitude = allocate_2d_array(ny, nx);

    // Initialize flow: zero velocity everywhere
    for (int j = 0; j < ny; j++) {
        for (int i = 0; i <= nx; i++) {
            h_u[j][i] = 0.0;
        }
    }
    for (int j = 0; j <= ny; j++) {
        for (int i = 0; i < nx; i++) {
            h_v[j][i] = 0.0;
        }
    }

    // Allocate device memory
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_u, ny*(nx+1)*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_v, (ny+1)*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_p, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_div, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_u_next, ny*(nx+1)*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_v_next, (ny+1)*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_u_center, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_v_center, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_velocity_magnitude, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMalloc((void **)&d_converged, s_i));

    // Initialize device memory
    CUDA_CHECK_ERROR(hipMemset(d_u, 0, ny*(nx+1)*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_v, 0, (ny+1)*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_p, 0, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_div, 0, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_u_next, 0, ny*(nx+1)*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_v_next, 0, (ny+1)*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_u_center, 0, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_v_center, 0, ny*nx*s_d));
    CUDA_CHECK_ERROR(hipMemset(d_velocity_magnitude, 0, ny*nx*s_d));

    // Copy data from host to device
    copy_host_to_device();

    // Define CUDA grid and block dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((nx + blockSize.x) / blockSize.x,
                  (ny + blockSize.y) / blockSize.y);

    // Open output files
    FILE *fp_u_center = fopen("u_center_data.bin", "wb");
    FILE *fp_v_center = fopen("v_center_data.bin", "wb");
    FILE *fp_magnitude = fopen("velocity_magnitude_data.bin", "wb");

    if (fp_u_center == NULL || fp_v_center == NULL || fp_magnitude == NULL) {
        perror("Error opening binary files");
        return 1;
    }

    int num_time_steps = (int)(total_time / dt);
    int output_interval = num_time_steps / max_number_of_frames;
    if (output_interval == 0) output_interval = 1;

    // Main time loop
    for (int t = 0; t < num_time_steps; t++) {
        clock_t step_start_time = clock();

        // Update velocities
        update_velocities_kernel<<<gridSize, blockSize>>>(
            d_u, d_v, d_p, d_u_next, d_v_next, nx, ny, dx, dy, dt, nu);
        // CUDA_CHECK_ERROR(cudaGetLastError());
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        // Copy updated velocities
        update_velocities_final_kernel<<<gridSize, blockSize>>>(
            d_u, d_v, d_u_next, d_v_next, nx, ny);
        // CUDA_CHECK_ERROR(cudaGetLastError());
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        // Pressure correction iterations
        for (int iter = 0; iter < max_iterations; iter++) {
            // Compute divergence
            compute_divergence_kernel<<<gridSize, blockSize>>>(d_u, d_v, d_div,
                                                               nx, ny, dx, dy);
            // CUDA_CHECK_ERROR(cudaGetLastError());
            CUDA_CHECK_ERROR(hipDeviceSynchronize());

            // Initialize convergence flag to 1
            int h_converged = 1;
            CUDA_CHECK_ERROR(hipMemcpy(d_converged, &h_converged, s_i,
                                        hipMemcpyHostToDevice));

            // Apply pressure correction (Red-Black)
            pressure_correction_kernel_red<<<gridSize, blockSize>>>(
                d_u, d_v, d_p, d_div, nx, ny, dx, dy, dt, beta,
                div_tolerance, d_converged);
            // CUDA_CHECK_ERROR(cudaGetLastError());
            CUDA_CHECK_ERROR(hipDeviceSynchronize());

            pressure_correction_kernel_black<<<gridSize, blockSize>>>(
                d_u, d_v, d_p, d_div, nx, ny, dx, dy, dt, beta,
                div_tolerance, d_converged);
            // CUDA_CHECK_ERROR(cudaGetLastError());
            CUDA_CHECK_ERROR(hipDeviceSynchronize());

            // Check if converged
            CUDA_CHECK_ERROR(hipMemcpy(&h_converged, d_converged, s_i,
                                        hipMemcpyDeviceToHost));

            if (h_converged) {
                break;
            }
        }

        // Apply boundary conditions
        apply_boundary_conditions_kernel<<<gridSize, blockSize>>>(
            d_u, d_v, nx, ny, inlet_start, inlet_end, outlet1_start, outlet1_end, outlet2_start, outlet2_end, outlet3_start, outlet3_end);
        // CUDA_CHECK_ERROR(cudaGetLastError());
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        // Calculate center velocities and magnitude
        calculate_center_velocities_and_magnitude_kernel<<<gridSize,
                                                           blockSize>>>(
            d_u, d_v, d_u_center, d_v_center, d_velocity_magnitude, nx, ny);
        // CUDA_CHECK_ERROR(cudaGetLastError());
        CUDA_CHECK_ERROR(hipDeviceSynchronize());

        clock_t step_end_time = clock();
        total_compute_time_seconds +=
            (double)(step_end_time - step_start_time) / CLOCKS_PER_SEC;

        // Output results at specified intervals
        if (t % output_interval == 0) {
            // Copy results from device to host for output
            copy_device_to_host();

            // Write to binary files
            write_state_to_binary(fp_u_center, fp_v_center, fp_magnitude);
        }
    }

    // Copy final results from device to host
    copy_device_to_host();

    // Close output files
    fclose(fp_u_center);
    fclose(fp_v_center);
    fclose(fp_magnitude);

    // Calculate total elapsed time
    clock_t total_end_time = clock();
    double total_time_seconds =
        (double)(total_end_time - total_start_time) / CLOCKS_PER_SEC;

    // Write simulation metadata
    write_metadata_to_json("simulation_metadata.json",
                           total_compute_time_seconds, total_time_seconds);

    // Free device memory
    hipFree(d_u);
    hipFree(d_v);
    hipFree(d_p);
    hipFree(d_div);
    hipFree(d_u_next);
    hipFree(d_v_next);
    hipFree(d_u_center);
    hipFree(d_v_center);
    hipFree(d_velocity_magnitude);
    hipFree(d_converged);

    // Free host memory
    free_2d_array(h_u);
    free_2d_array(h_v);
    free_2d_array(h_p);
    free_2d_array(h_div);
    free_2d_array(h_u_center);
    free_2d_array(h_v_center);
    free_2d_array(h_velocity_magnitude);

    return 0;
}
